#include "hip/hip_runtime.h"
#include "reduce_by_key.cuh"

#define TB_SIZE 256
#define MAX_K 5

using std::cout;
using std::endl;
using std::vector;

void print(float3 point) {
    cout << "float3(" << point.x << ", " << point.y << ", " << point.z << ") ";
}

void println(float3 point) {
    cout << "float3(" << point.x << ", " << point.y << ", " << point.z << ")" << endl;
}

__device__ __host__ float3 operator+(const float3 &a, const float3 &b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

inline __host__ __device__ void operator+=(float3 &a, float3 b)
{
    a.x += b.x; a.y += b.y; a.z += b.z;
}

__global__ void my_reduce_by_key_kernel(int n, int k, int *keys, float3 *values, float3 *almost_reduced_values) {
    __shared__ float3 partial_sum[TB_SIZE][MAX_K];

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) {
        return;
    }
    // Initalize shared memory to zero!
    for (int i = 0; i < k; ++i) {
        partial_sum[threadIdx.x][i] = make_float3(0, 0, 0);
    }
    const int key = keys[tid];  // value from 0 to k-1
    // Load elements into shared memory
    partial_sum[threadIdx.x][key] = values[tid];
    __syncthreads();

    for (int s = 1; s < blockDim.x; s <<= 1) {
        if (threadIdx.x % (2*s) == 0) {
            for (int i = 0; i < k; ++i) {
                partial_sum[threadIdx.x][i] += partial_sum[threadIdx.x + s][i];
            }
        }
        __syncthreads();
    }

    // Frist thread in a block writes to main memory
    if (threadIdx.x == 0) {
        for (int i = 0; i < k; ++i) {
            const int pos = blockIdx.x * k + i;
            almost_reduced_values[pos] = partial_sum[0][i];
        }
    }
}

// run at the end of the reduce by key with only one block
__global__ void sum_reduce(int n, int k, float3 *d_almost_reduces_values, float3 *output) {
    __shared__ float3 partial_sum[TB_SIZE][MAX_K];
    const int tid = threadIdx.x;
    for (int i = 0; i < k; ++i) {
        const int pos = tid * k + i;
        partial_sum[tid][i] = d_almost_reduces_values[pos];
    }
    __syncthreads();
    for (int s = 1; s < blockDim.x; s <<= 1) {
        if (tid % (2*s) == 0) {
            for (int i = 0; i < k; ++i) {
                partial_sum[tid][i] += partial_sum[threadIdx.x + s][i];
            }
        }
        __syncthreads();
    }
    if (tid == 0) {
        for (int i = 0; i < k; ++i) {
            output[i] = partial_sum[0][i];
        }
    }
}

void my_reduce_by_key(int n, int k, int *d_keys, 
                      float3* d_values, 
                      float3 *d_almost_reduced_values, 
                      float3 *d_output) {
    const int N_BLOCKS = (n + TB_SIZE - 1) / TB_SIZE;
    my_reduce_by_key_kernel<<<N_BLOCKS, TB_SIZE>>> (n, k, d_keys, d_values, d_almost_reduced_values);
    // if (n > TB_SIZE) 
    sum_reduce<<<1, TB_SIZE>>> (N_BLOCKS, k, d_almost_reduced_values, d_output);
}
